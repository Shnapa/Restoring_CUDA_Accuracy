#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

int main(){

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(0.0f, 1001.0f); // Generates numbers in [0.0, 1.0)


    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    const int m = 1000, n = 1000, k = 1000;
    // __half h_A[m * k] = {__float2half(1.0f), __float2half(2.0f), __float2half(3.0f), __float2half(4.0f)};
    // __half h_B[k * n] = {__float2half(5.0f), __float2half(6.0f), __float2half(7.0f), __float2half(8.0f)};

    __half h_A[m * k];
    __half h_B[k * n];
    float h_C[m * n]; // accumulator / result
    for (int i = 0; i < m*k; i++) {
        h_A[i] = __half2float(dist(gen));
        h_B[i] = __half2float(dist(gen));
        h_C[i] = 0.0f;
    }

    __half *d_A, *d_B;
    float *d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(__half));
    hipMalloc((void**)&d_B, k * n * sizeof(__half));
    hipMalloc((void**)&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, h_A, m * k * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, m * n * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    // C = α × (A × B) + β × C
    status = hipblasGemmEx(handle,
                         HIPBLAS_OP_N, HIPBLAS_OP_N, // transpose matrices or no, N - no
                         m, n, k,
                         &alpha,
                         d_A,
                         HIP_R_16F, // FP16
                         m, // number of rows of A
                         d_B,
                         HIP_R_16F, // FP16
                         k, // number of rows of B
                         &beta,
                         d_C,
                         HIP_R_32F, // FP32(float)
                         m, // number of rows of C
                         HIP_R_32F, // FP32(float)
                         HIPBLAS_GEMM_DEFAULT);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        return -1;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuBLAS matrix multiplication took: %f ms\n", milliseconds);

    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // printf("Result matrix C:\n");
    // for (int i = 0; i < m * n; i++) {
    //     printf("%.2f ", h_C[i]);
    //     if ((i + 1) % n == 0) printf("\n");
    // }

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

}
