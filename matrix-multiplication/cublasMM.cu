#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "mmul.cuh"

void cublasMatrixMultiply(const float* h_A, const float* h_B, float* h_C,
                          const size_t m, const size_t k, const size_t n,
                          float& executionTime) {
    const size_t size_A = m * k * sizeof(float);
    const size_t size_B = k * n * sizeof(float);
    const size_t size_C = m * n * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemset(d_C, 0, size_C);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    constexpr float alpha = 1.0f;
    constexpr float beta = 0.0f;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipblasGemmEx(handle,
               HIPBLAS_OP_N, HIPBLAS_OP_N,
               n, m, k,
               &alpha,
               d_B, HIP_R_32F, n,
               d_A, HIP_R_32F, k,
               &beta,
               d_C, HIP_R_32F, n,
               HIPBLAS_COMPUTE_32F,
               HIPBLAS_GEMM_DEFAULT);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&executionTime, start, stop);
    hipDeviceSynchronize();
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void cublasMatrixMultiplyHalf(const __half* h_A, const __half* h_B, float* h_C,
                          const size_t m, const size_t k, const size_t n,
                          float& executionTime) {
    const size_t size_A = m * k * sizeof(__half);
    const size_t size_B = k * n * sizeof(__half);
    const size_t size_C = m * n * sizeof(float);

    __half *d_A, *d_B;
    float *d_C;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemset(d_C, 0, size_C);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    constexpr float alpha = 1.0f;
    constexpr float beta = 0.0f;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipblasGemmEx(handle,
               HIPBLAS_OP_N, HIPBLAS_OP_N,
               n, m, k,
               &alpha,
               d_B, HIP_R_16F, n,
               d_A, HIP_R_16F, k,
               &beta,
               d_C, HIP_R_32F, n,
               HIPBLAS_COMPUTE_32F,
               HIPBLAS_GEMM_DEFAULT);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&executionTime, start, stop);

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
