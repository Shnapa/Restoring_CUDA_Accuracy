#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdlib>
#include "../include/matrixParser.h"
#include <benchmark/benchmark.h>

using namespace nvcuda;

#define TILE_SIZE 16
#define WARP_SIZE 32

void loadMatrices_RR_half(const std::string &filePath,
                     std::vector<__half> &A,
                     std::vector<__half> &B)
{
    size_t m, k, n;
    parseDimensions(filePath, m, k, n);
    std::ifstream fin(filePath);
    if (!fin.is_open()) std::exit(EXIT_FAILURE);
    std::string line;
    std::getline(fin, line);
    std::istringstream isa(line);
    float v;
    size_t i = 0;
    while (isa >> v && i < A.size()) A[i++] = __float2half(v);
    std::getline(fin, line);
    isa.clear();
    isa.str(line);
    i = 0;
    while (isa >> v && i < B.size()) B[i++] = __float2half(v);
}

__global__ void matrixMultiplyWMMA(const half* A, const half* B, float* C,
const size_t padded_M, const size_t padded_N, const size_t padded_K) {
    const size_t warpM = blockIdx.x;
    const size_t warpN = blockIdx.y;

    if (warpM * TILE_SIZE >= padded_M || warpN * TILE_SIZE >= padded_N)
        return;

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TILE_SIZE, TILE_SIZE, TILE_SIZE, __half, nvcuda::wmma::row_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TILE_SIZE, TILE_SIZE, TILE_SIZE, __half, nvcuda::wmma::row_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TILE_SIZE, TILE_SIZE, TILE_SIZE, float> acc_frag;

    fill_fragment(acc_frag, 0.0f);

    for (size_t tileK = 0; tileK < padded_K; tileK += TILE_SIZE) {
        const half* tileA = A + warpM * TILE_SIZE * padded_K + tileK;
        const half* tileB = B + tileK * padded_N + warpN * TILE_SIZE;

        load_matrix_sync(a_frag, tileA, padded_K);
        load_matrix_sync(b_frag, tileB, padded_N);
        mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }

    float* tileC = C + warpM * TILE_SIZE * padded_N + warpN * TILE_SIZE;
    store_matrix_sync(tileC, acc_frag, padded_N, nvcuda::wmma::mem_row_major);
}

static void BM_RunMultiplicationWMMA(benchmark::State &state, const std::string &filePath) {
    size_t m, k, n;
    parseDimensions(filePath, m, k, n);

    std::vector<float> h_A(m*k), h_B(n*k), h_C(m*n, 0.0f);
    loadMatrices_RR(filePath, h_A, h_B);

    const size_t padded_M = (m + TILE_SIZE - 1) / TILE_SIZE * TILE_SIZE;
    const size_t padded_K = (k + TILE_SIZE - 1) / TILE_SIZE * TILE_SIZE;
    const size_t padded_N = (n + TILE_SIZE - 1) / TILE_SIZE * TILE_SIZE;

    const auto h_A_pad = new __half[padded_M * padded_K];
    const auto h_B_pad = new __half[padded_K * padded_N];
    const auto h_C_pad = new float[padded_M * padded_N];

    for (size_t i = 0; i < padded_M; i++) {
        for (size_t j = 0; j < padded_K; j++) {
            if (i < m && j < k) {
                h_A_pad[i * padded_K + j] = __float2half(h_A[i * k + j]);
            } else {
                h_A_pad[i * padded_K + j] = __float2half(0.0f);
            }
        }
    }

    for (size_t i = 0; i < padded_K; i++) {
        for (size_t j = 0; j < padded_N; j++) {
            if (i < k && j < n) {
                h_B_pad[i * padded_N + j] = __float2half(h_B[i * n + j]);
            } else {
                h_B_pad[i * padded_N + j] = __float2half(0.0f);
            }
        }
    }

    for (size_t i = 0; i < padded_M * padded_N; i++) {
        h_C_pad[i] = 0.0f;
    }

    __half *d_A, *d_B;
    float *d_C;
    hipMalloc(&d_A, padded_M * padded_K * sizeof(__half));
    hipMalloc(&d_B, padded_K * padded_N * sizeof(__half));
    hipMalloc(&d_C, padded_M * padded_N * sizeof(float));

    hipMemcpy(d_A, h_A_pad, padded_M * padded_K * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B_pad, padded_K * padded_N * sizeof(__half), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, padded_M * padded_N * sizeof(float));

    dim3 threads(WARP_SIZE, 1);
    dim3 blocks(padded_M / TILE_SIZE, padded_N / TILE_SIZE);

    for (auto _ : state) {
        hipMemset(d_C, 0, padded_M * padded_N * sizeof(float));
        matrixMultiplyWMMA<<<blocks, threads>>>(d_A, d_B, d_C, padded_M, padded_N, padded_K);
        hipDeviceSynchronize();
        benchmark::ClobberMemory();
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A_pad;
    delete[] h_B_pad;
    delete[] h_C_pad;
}

int main(int argc, char** argv) {
    for (const auto &filepath : filePaths) {
        benchmark::RegisterBenchmark(filepath, [filepath](benchmark::State &state) {
            BM_RunMultiplicationWMMA(state, filepath);
        });
    }
    benchmark::Initialize(&argc, argv);
    benchmark::RunSpecifiedBenchmarks();
    return 0;
}
