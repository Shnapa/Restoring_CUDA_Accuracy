#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "matrixParser.h"
#include <benchmark/benchmark.h>
#include <fstream>
#include <vector>
#include <string>

int loadHalfMatricesFromFileArray(const std::string &filePath, __half* A, size_t A_elements, __half* B, size_t B_elements) {
    std::ifstream file(filePath);
    std::string line;
    std::istringstream issA(line);
    size_t countA = 0;
    float value;
    while (issA >> value) {
        if (countA < A_elements) {
            A[countA++] = __float2half(value);
        } else {
            break;
        }
    }
    std::istringstream issB(line);
    size_t countB = 0;
    while (issB >> value) {
        if (countB < B_elements) {
            B[countB++] = __float2half(value);
        } else {
            break;
        }
    }
    return 0;
}

static void BM_cublasMul(benchmark::State& state, const std::string &filePath) {
    size_t m, n, k;
    parseDimensions(filePath, m, n, k);
    const size_t A_elements = m * n;
    const size_t B_elements = n * k;
    const size_t C_elements = m * k;
    auto* h_A = static_cast<__half*>(malloc(A_elements * sizeof(__half)));
    auto* h_B = static_cast<__half*>(malloc(B_elements * sizeof(__half)));
    auto* h_C = static_cast<float*>(malloc(C_elements * sizeof(float)));

    loadHalfMatricesFromFileArray(filePath, h_A, A_elements, h_B, B_elements);

    __half *d_A, *d_B;
    float *d_C;

    hipMalloc(&d_A, A_elements * sizeof(__half));
    hipMalloc(&d_B, B_elements * sizeof(__half));
    hipMalloc(&d_C, C_elements * sizeof(float));

    hipMemcpy(d_A, h_A, A_elements * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, B_elements * sizeof(__half), hipMemcpyHostToDevice);

    free(h_A);
    free(h_B);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    constexpr float alpha = 1.0f;
    constexpr float beta = 0.0f;

    for (auto _ : state) {
        hipMemset(d_C, 0, C_elements * sizeof(float));
        hipblasGemmEx(handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m, k, n,
            &alpha,
            d_A, HIP_R_16F, m,
            d_B, HIP_R_16F, n,
            &beta,
            d_C, HIP_R_32F, m,
            HIP_R_32F,
            HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
    }

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_C);
}

int main(int argc, char** argv) {
    for (const auto &filepath : filePaths) {
        benchmark::RegisterBenchmark(filepath, [filepath](benchmark::State &state) {
            BM_cublasMul(state, filepath);
        });
    }
    benchmark::Initialize(&argc, argv);
    benchmark::RunSpecifiedBenchmarks();
    return 0;
}