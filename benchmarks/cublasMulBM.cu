#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "../include/matrixParser.h"
#include <benchmark/benchmark.h>
#include <fstream>
#include <vector>
#include <string>

int loadHalfMatricesFromFileArray(const std::string &filePath, __half* A, size_t A_elements, __half* B, size_t B_elements) {
    std::ifstream file(filePath);
    std::string line;
    std::getline(file, line);
    std::istringstream issA(line);
    size_t countA = 0;
    float value;
    while (issA >> value && countA < A_elements) {
        A[countA++] = __float2half(value);
    }
    std::getline(file, line);
    std::istringstream issB(line);
    size_t countB = 0;
    while (issB >> value && countB < B_elements) {
        B[countB++] = __float2half(value);
    }
    return 0;
}

static void BM_cublasMul(benchmark::State& state, const std::string &filePath) {
    size_t m, k, n;
    parseDimensions(filePath, m, k, n);
    const size_t sizeA = m * k;
    const size_t sizeB = k * n;
    const size_t sizeC = m * n;

    std::vector<float> h_A(sizeA), h_B(sizeB), h_C(sizeC);
    loadMatrices_CC(filePath, h_A, h_B);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA * sizeof(float));
    hipMalloc(&d_B, sizeB * sizeof(float));
    hipMalloc(&d_C, sizeC * sizeof(float));

    hipMemcpy(d_A, h_A.data(), sizeA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), sizeB * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, sizeC * sizeof(float));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    constexpr float alpha = 1.0f, beta = 0.0f;

    for (auto _ : state) {
        hipblasGemmEx(handle,
                 HIPBLAS_OP_T, HIPBLAS_OP_T,
                 n, m, k,
                 &alpha,
                 d_B, HIP_R_32F, k,
                 d_A, HIP_R_32F, m,
                 &beta,
                 d_C, HIP_R_32F, n,
                 HIP_R_32F,
                 HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
    }
    hipMemcpy(h_C.data(), d_C, sizeC * sizeof(float), hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

int main(int argc, char** argv) {
    for (const auto &filepath : filePaths) {
        benchmark::RegisterBenchmark(filepath, [filepath](benchmark::State &state) {
            BM_cublasMul(state, filepath);
        });
    }
    benchmark::Initialize(&argc, argv);
    benchmark::RunSpecifiedBenchmarks();
    return 0;
}
