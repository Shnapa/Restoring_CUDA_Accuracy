#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <chrono>

#define TILE_SIZE 32

__global__ void matrixMultiplyTiled(float *A, float *B, float *C, int N) {
    // Спільна пам’ять для тайлів
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    // Індекси поточного потоку
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    // Кількість підматриць, які потрібно обробити
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Завантажуємо дані в спільну пам’ять (перевіряємо межі)
        if (row < N && (t * TILE_SIZE + threadIdx.x) < N)
            tileA[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && (t * TILE_SIZE + threadIdx.y) < N)
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads(); // Синхронізація потоків

        // Виконуємо множення для даного тайлу
        for (int i = 0; i < TILE_SIZE; i++)
            sum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];

        __syncthreads(); // Очікуємо завершення перед наступною ітерацією
    }

    // Записуємо результат у глобальну пам’ять
    if (row < N && col < N)
        C[row * N + col] = sum;
}

void multiplyMatrices(float *h_A, float *h_B, float *h_C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    matrixMultiplyTiled<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void writeExecutionTimeToFile(double executionTime, int N, const std::string& filename) {
    std::ofstream file(filename, std::ios::app);

    if (file.is_open()) {
        if (file.tellp() == 0) {
            file << "Size, Execution Time (seconds)\n";
        }
        file << N << ", " << executionTime << "\n";
        file.close();
    } else {
        std::cerr << "Error opening file for writing execution time!" << std::endl;
    }
}

int main() {
    int sizes[] = {10, 100, 1000, 10000};
    size_t size;

    for (int i = 0; i < 3; i++) {
        int N = sizes[i];
        size = N * N * sizeof(float);

        float *h_A = (float*)malloc(size);
        float *h_B = (float*)malloc(size);
        float *h_C = (float*)malloc(size);

        for (int j = 0; j < N * N; j++) {
            h_A[j] = 1.0f;
            h_B[j] = 1.0f;
        }

        auto start = std::chrono::high_resolution_clock::now();
        multiplyMatrices(h_A, h_B, h_C, N);
        auto end = std::chrono::high_resolution_clock::now();

        std::chrono::duration<double> duration = end - start;
        writeExecutionTimeToFile(duration.count(), N, "execution_time.txt");

        std::cout << "C[0][0] = " << h_C[0] << std::endl;

        free(h_A);
        free(h_B);
        free(h_C);
    }

    return 0;
}

