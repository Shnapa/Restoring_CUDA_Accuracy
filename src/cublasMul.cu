#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "timeMeasurement.h"

int main(){
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(1.0f, 10000.0f);

    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    const int m = 1000, n = 1000, k = 1000;

    __half* h_A = static_cast<__half*>(malloc(m * k * sizeof(__half)));
    __half* h_B = static_cast<__half*>(malloc(k * n * sizeof(__half)));
    float*  h_C = static_cast<float*>(malloc(m * n * sizeof(float)));

    for (int i = 0; i < m*k; i++) {
        h_A[i] = __half2float(dist(gen));
        h_B[i] = __half2float(dist(gen));
        h_C[i] = 0.0f;
    }

    __half *d_A, *d_B;
    float *d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(__half));
    hipMalloc((void**)&d_B, k * n * sizeof(__half));
    hipMalloc((void**)&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, h_A, m * k * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, m * n * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // C = α × (A × B) + β × C
    status = hipblasGemmEx(handle,
                         HIPBLAS_OP_N, HIPBLAS_OP_N, // transpose matrices or no, N - no
                         m, n, k,
                         &alpha,
                         d_A,
                         HIP_R_16F, // FP16
                         m, // number of rows of A
                         d_B,
                         HIP_R_16F, // FP16
                         k, // number of rows of B
                         &beta,
                         d_C,
                         HIP_R_32F, // FP32(float)
                         m, // number of rows of C
                         HIP_R_32F, // FP32(float)
                         HIPBLAS_GEMM_DEFAULT);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        return -1;
    }

    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

}
