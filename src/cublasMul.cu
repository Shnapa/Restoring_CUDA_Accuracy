#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "matrixParser.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "timeMeasurement.h"

int loadHalfMatricesFromFileArray(const std::string &filePath, __half* A, size_t A_elements, __half* B, size_t B_elements) {
    std::ifstream file(filePath);
    std::string line;
    std::getline(file, line);
    std::istringstream issA(line);
    size_t countA = 0;
    float value;
    while (issA >> value && countA < A_elements) {
        A[countA++] = __float2half(value);
    }
    std::getline(file, line);
    std::istringstream issB(line);
    size_t countB = 0;
    while (issB >> value && countB < B_elements) {
        B[countB++] = __float2half(value);
    }
    return 0;
}
inline bool compareFloats(float a, float b, float epsilon = 1e-7) {
    float res = std::abs((b - a)/a);
    return res < epsilon;
}
void compare(const float* h_C, size_t m, size_t n, size_t k, const std::string& filePath) {
    size_t A_elements = m * n;
    size_t B_elements = n * k;

    // Allocate and reload host matrices A and B
    auto* A = static_cast<float*>(malloc(A_elements * sizeof(float)));
    auto* B = static_cast<float*>(malloc(B_elements * sizeof(float)));
    loadMatricesFromFileArray(filePath, A, A_elements, B, B_elements);

    // Allocate memory for CPU result
    auto* C_cpu = static_cast<float*>(malloc(m * k * sizeof(float)));

    // Brute-force matrix multiplication
    for (size_t i = 0; i < m; ++i) {
        for (size_t j = 0; j < k; ++j) {
            float sum = 0.0f;
            for (size_t l = 0; l < n; ++l) {
                sum += A[i * n + l] * B[l * k + j];
            }
            C_cpu[i * k + j] = sum;
        }
    }

    // Compare CPU result with GPU result using epsilon for floating point comparison
    bool match = true;
    const float epsilon = 1e-5;
    for (size_t i = 0; i < m * k; ++i) {
        if (!compareFloats(C_cpu[i], h_C[i], epsilon)) {
            std::cerr << "Mismatch at index " << i << ": CPU = " << C_cpu[i]
                      << ", GPU = " << h_C[i] << std::endl;
            match = false;
            break;
        }
    }

    if (match) {
        std::cout << "Verification passed: CPU and GPU results match." << std::endl;
    } else {
        std::cout << "Verification failed: CPU and GPU results do not match." << std::endl;
    }

    free(A);
    free(B);
    free(C_cpu);
}
int main(int argc, char** argv) {
    if(argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_file_path>" << std::endl;
        return 1;
    }
    const std::string filePath = argv[1];

    size_t m, n, k;
    parseDimensions(filePath, m, n, k);
    const size_t A_elements = m * n;
    const size_t B_elements = n * k;
    const size_t C_elements = m * k;
    auto* h_A = static_cast<float*>(malloc(A_elements * sizeof(float)));
    auto* h_B = static_cast<float*>(malloc(B_elements * sizeof(float)));
    auto* h_C = static_cast<float*>(malloc(C_elements * sizeof(float)));

    loadMatricesFromFileArray(filePath, h_A, A_elements, h_B, B_elements);

    float *d_A, *d_B;
    float *d_C;

    hipMalloc(&d_A, A_elements * sizeof(float));
    hipMalloc(&d_B, B_elements * sizeof(float));
    hipMalloc(&d_C, C_elements * sizeof(float));

    hipMemcpy(d_A, h_A, A_elements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, B_elements * sizeof(float), hipMemcpyHostToDevice);

    free(h_A);
    free(h_B);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    constexpr float alpha = 1.0f;
    constexpr float beta = 0.0f;


    hipMemset(d_C, 0, C_elements * sizeof(float));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipblasGemmEx(   handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                    m, k, n,
                    &alpha,
                    d_A, HIP_R_32F, m,
                    d_B, HIP_R_32F, n,
                    &beta,
                    d_C, HIP_R_32F, m,
                    HIP_R_32F,
                    HIPBLAS_GEMM_DEFAULT);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time elapsed (ms): " << elapsedTime << std::endl;

    hipMemcpy(h_C, d_C, C_elements * sizeof(float), hipMemcpyDeviceToHost);
    compare(h_C, m, n, k, filePath);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_C);
}
