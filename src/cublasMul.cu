#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "matrixParser.h"
#include "timeMeasurement.h"
#include "compare.cu"

int main(const int argc, char** argv) {
    if (argc < 2) return 1;
    const std::string filePath = argv[1];

    int m, k, n;
    parseDimensions(filePath, m, k, n);

    const int size_A = m * k;
    const int size_B = k * n;
    const int size_C = m * n;

    std::vector<float> h_A(size_A), h_B(size_B), h_C(size_C);
    loadMatrices_CC(filePath, h_A, h_B);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_A * sizeof(float));
    hipMalloc(&d_B, size_B * sizeof(float));
    hipMalloc(&d_C, size_C * sizeof(float));

    hipMemcpy(d_A, h_A.data(), size_A * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size_B * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, size_C * sizeof(float));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    constexpr float alpha = 1.0f, beta = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, nullptr);

    hipblasGemmEx(handle,
                 HIPBLAS_OP_T, HIPBLAS_OP_T,
                 n, m, k,
                 &alpha,
                 d_B, HIP_R_32F, k,
                 d_A, HIP_R_32F, m,
                 &beta,
                 d_C, HIP_R_32F, n,
                 HIP_R_32F,
                 HIPBLAS_GEMM_DEFAULT);

    hipDeviceSynchronize();
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);

    float elapsed = 0.0f;
    hipEventElapsedTime(&elapsed, start, stop);
    std::cout << elapsed << "\n";

    hipMemcpy(h_C.data(), d_C, size_C * sizeof(float), hipMemcpyDeviceToHost);

    compare(h_C, m, k, n, filePath);

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}