#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "matrixParser.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "timeMeasurement.h"

int loadHalfMatricesFromFileArray(const std::string &filePath, __half* A, size_t A_elements, __half* B, size_t B_elements) {
    std::ifstream file(filePath);
    std::string line;
    std::getline(file, line);
    std::istringstream issA(line);
    size_t countA = 0;
    float value;
    while (issA >> value && countA < A_elements) {
        A[countA++] = __float2half(value);
    }
    std::getline(file, line);
    std::istringstream issB(line);
    size_t countB = 0;
    while (issB >> value && countB < B_elements) {
        B[countB++] = __float2half(value);
    }
    return 0;
}

int main(int argc, char** argv) {
    if(argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_file_path>" << std::endl;
        return 1;
    }
    const std::string filePath = argv[1];

    size_t m, n, k;
    parseDimensions(filePath, m, n, k);
    const size_t A_elements = m * n;
    const size_t B_elements = n * k;
    const size_t C_elements = m * k;
    auto* h_A = static_cast<float*>(malloc(A_elements * sizeof(float)));
    auto* h_B = static_cast<float*>(malloc(B_elements * sizeof(float)));
    auto* h_C = static_cast<float*>(malloc(C_elements * sizeof(float)));

    loadMatricesFromFileArray(filePath, h_A, A_elements, h_B, B_elements);

    float *d_A, *d_B;
    float *d_C;

    hipMalloc(&d_A, A_elements * sizeof(float));
    hipMalloc(&d_B, B_elements * sizeof(float));
    hipMalloc(&d_C, C_elements * sizeof(float));

    hipMemcpy(d_A, h_A, A_elements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, B_elements * sizeof(float), hipMemcpyHostToDevice);

    free(h_A);
    free(h_B);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    constexpr float alpha = 1.0f;
    constexpr float beta = 0.0f;


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMemset(d_C, 0, C_elements * sizeof(float));
    hipblasGemmEx(   handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                    m, k, n,
                    &alpha,
                    d_A, HIP_R_32F, m,
                    d_B, HIP_R_32F, n,
                    &beta,
                    d_C, HIP_R_32F, m,
                    HIP_R_32F,
                    HIPBLAS_GEMM_DEFAULT);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime = 0;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time elapsed (ms): " << elapsedTime << std::endl;

    hipMemcpy(h_C, d_C, C_elements * sizeof(float), hipMemcpyDeviceToHost);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_C);
}
