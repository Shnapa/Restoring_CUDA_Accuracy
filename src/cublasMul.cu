#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "matrixParser.h"
#include "timeMeasurement.h"
#include "compare.cu"

int main(int argc, char** argv) {
    if (argc < 2) return 1;
    std::string filePath = argv[1];

    size_t m, k, n;
    parseDimensions(filePath, m, k, n);

    const size_t sizeA = m * k;
    const size_t sizeB = k * n;
    const size_t sizeC = m * n;

    std::vector<float> h_A(sizeA), h_B(sizeB), h_C(sizeC);
    loadMatrices_CC(filePath, h_A, h_B);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA * sizeof(float));
    hipMalloc(&d_B, sizeB * sizeof(float));
    hipMalloc(&d_C, sizeC * sizeof(float));

    hipMemcpy(d_A, h_A.data(), sizeA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), sizeB * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, sizeC * sizeof(float));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    constexpr float alpha = 1.0f, beta = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, nullptr);

    hipblasGemmEx(handle,
                 HIPBLAS_OP_T, HIPBLAS_OP_T,
                 n, m, k,
                 &alpha,
                 d_B, HIP_R_32F, k,
                 d_A, HIP_R_32F, m,
                 &beta,
                 d_C, HIP_R_32F, n,
                 HIP_R_32F,
                 HIPBLAS_GEMM_DEFAULT);

    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);

    float elapsed = 0.0f;
    hipEventElapsedTime(&elapsed, start, stop);
    std::cout << elapsed << "\n";

    hipMemcpy(h_C.data(), d_C, sizeC * sizeof(float), hipMemcpyDeviceToHost);

    compare(h_C, m, k, n, filePath);

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}