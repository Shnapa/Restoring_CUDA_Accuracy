#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "timeMeasurement.h"
#include "matrixParser.h"

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <A_matrix_path> <B_matrix_path> "<< std::endl;
        return 1;
    }

    std::string A_matrix_path = argv[1];
    std::string B_matrix_path = argv[2];

    int m, n, k;

    float* A = loadMatrixFromFileToArray(A_matrix_path, m, n);
    float* B = loadMatrixFromFileToArray(B_matrix_path, n, k);

    __half* h_A = static_cast<__half*>(malloc(m * n * sizeof(__half)));
    __half* h_B = static_cast<__half*>(malloc(n * k * sizeof(__half)));
    float* h_C = static_cast<float*>(malloc(m * k * sizeof(float)));

    for (int i = 0; i < m * n; i++) {
        h_A[i] = __float2half(A[i]);
    }

    for (int i = 0; i < n * k; i++) {
        h_B[i] = __float2half(B[i]);
    }

    __half *d_A, *d_B;
    float *d_C;
    size_t sizeA = m * n * sizeof(__half);
    size_t sizeB = n * k * sizeof(__half);
    size_t sizeC = m * k * sizeof(float);
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, sizeC, hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    auto start  = get_current_time_fenced();

    hipblasStatus_t status = hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        m, k, n,
        &alpha,
        d_A, HIP_R_16F, m,
        d_B, HIP_R_16F, n,
        &beta,
        d_C, HIP_R_32F, m,
        HIP_R_32F, HIPBLAS_GEMM_DEFAULT
    );

    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CUBLAS gemm failed!" << std::endl;
        return -1;
    }

    auto end  = get_current_time_fenced();
    std::cout << "Elapsed time: " << to_ms(end-start) << " ms" << std::endl;

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

