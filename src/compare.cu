#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_fp16.h>
#include <iomanip>

#include "../include/matrixParser.h"

bool compareFloats(const float a, const float b, const float epsilon) {
    const float res = std::abs((b - a)/a);
    return res < epsilon;
}

void compare(const std::vector<float>& h_C,
             const size_t m, const size_t k, const size_t n,
             const std::string& filePath) {
    const size_t size_A = m * k;
    const size_t size_B = k * n;
    const size_t size_C = m * n;

    std::vector<float> A(size_A), B(size_B), C_cpu(size_C);

    loadMatrices_RR(filePath, A, B);

    for (size_t i = 0; i < m; ++i) {
        for (size_t j = 0; j < n; ++j) {
            float sum = 0.0f;
            for (size_t l = 0; l < k; ++l) {
                sum += A[i * k + l] * B[l * n + j];
                C_cpu[i * n + j] = sum;
            }
        }
    }

    constexpr std::array<float,8> eps = {
        1e-5f, 1e-6f, 1e-7f, 1e-8f,
        1e-9f, 1e-10f, 1e-11f, 1e-12f
    };

    std::vector<size_t> firstMismatch(eps.size(), size_C);

    for (size_t i = 0; i < size_C; ++i) {
        float cpu = C_cpu[i];
        float gpu = h_C[i];
        for (size_t j = 0; j < eps.size(); ++j) {
            if (firstMismatch[j] == size_C && !compareFloats(cpu, gpu, eps[j])) {
                firstMismatch[j] = i;
            }
        }
        bool all_failed = true;
        for (size_t idx : firstMismatch) {
            if (idx == size_C) {
                all_failed = false;
                break;
            }
        }
        if (all_failed) break;
    }

    for (size_t j = 0; j < eps.size(); ++j) {
        float e = eps[j];
        if (firstMismatch[j] == size_C) {
            std::cout << "PASS @ eps=" << std::scientific << e << "\n";
        } else {
            size_t idx = firstMismatch[j];
            std::cerr << std::fixed << std::setprecision(12)
                      << "FAIL @ eps=" << e
                      << " at idx=" << idx
                      << ": CPU=" << C_cpu[idx]
                      << " GPU=" << h_C[idx] << "\n";
        }
    }
}

void loadMatrices_RR_half(const std::string &filePath,
                     std::vector<__half> &A,
                     std::vector<__half> &B)
{
    size_t m, k, n;
    parseDimensions(filePath, m, k, n);
    std::ifstream fin(filePath);
    if (!fin.is_open()) std::exit(EXIT_FAILURE);
    std::string line;
    std::getline(fin, line);
    std::istringstream isa(line);
    float v;
    size_t i = 0;
    while (isa >> v && i < A.size()) A[i++] = __float2half(v);
    std::getline(fin, line);
    isa.clear();
    isa.str(line);
    i = 0;
    while (isa >> v && i < B.size()) B[i++] = __float2half(v);
}

// void compare_half(const std::vector<float>& h_C,
//              const size_t m, const size_t k, const size_t n,
//              const std::string& filePath)
// {
//     const size_t size_A = m * k;
//     const size_t size_B = k * n;
//     const size_t size_C = m * n;
//
//     std::vector<__half> A(size_A), B(size_B);
//     std::vector<float> C(size_C);
//
//     loadMatrices_RR_half(filePath, A, B);
//
//     float time = 0.0f;
//     cublasMatrixMultiply(A.data(), B.data(), C.data(), m, k, n, time);
//
//     constexpr std::array<float,8> eps = {
//         1e-5f, 1e-6f, 1e-7f, 1e-8f,
//         1e-9f, 1e-10f, 1e-11f, 1e-12f
//     };
//
//     std::vector firstMismatch(eps.size(), size_C);
//
//     for (size_t i = 0; i < size_C; ++i) {
//         const float cpu = C[i];
//         const float gpu = h_C[i];
//         for (size_t j = 0; j < eps.size(); ++j) {
//             if (firstMismatch[j] == size_C && !compareFloats(cpu, gpu, eps[j])) {
//                 firstMismatch[j] = i;
//             }
//         }
//         bool all_failed = true;
//         for (const size_t idx : firstMismatch) {
//             if (idx == size_C) {
//                 all_failed = false;
//                 break;
//             }
//         }
//         if (all_failed) break;
//     }
//
//     for (size_t j = 0; j < eps.size(); ++j) {
//         const float e = eps[j];
//         if (firstMismatch[j] == size_C) {
//             std::cout << "PASS @ eps=" << std::scientific << e << "\n";
//         } else {
//             const size_t idx = firstMismatch[j];
//             std::cerr << std::fixed
//                       << "FAIL @ eps=" << e
//                       << " at idx=" << idx
//                       << ": CPU=" << C[idx]
//                       << " GPU=" << h_C[idx] << "\n";
//         }
//     }
// }