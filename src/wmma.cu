#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdlib>
#include "matrixParser.h"

using namespace nvcuda;

#define M 16
#define N 16
#define K 16
#define WMMA_N 32

int loadHalfMatricesFromFileArray(const std::string &filePath, __half* A, size_t A_elements, __half* B, size_t B_elements) {
    std::ifstream file(filePath);
    std::string line;
    std::getline(file, line);
    std::istringstream issA(line);
    size_t countA = 0;
    float value;
    while (issA >> value && countA < A_elements) {
        A[countA++] = __float2half(value);
    }
    std::getline(file, line);
    std::istringstream issB(line);
    size_t countB = 0;
    while (issB >> value && countB < B_elements) {
        B[countB++] = __float2half(value);
    }
    return 0;
}

__global__ void matrixMultiplyWMMA(const __half *A, const __half *B, float *C, size_t m, size_t n, size_t k) {
    const size_t warpM = (blockIdx.y * blockDim.y + threadIdx.y) * M;
    const size_t warpN = (blockIdx.x * blockDim.x + threadIdx.x) * N;

    if (warpM >= m || warpN >= n) return;

    wmma::fragment<wmma::matrix_a, M, N, K, __half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, M, N, K, __half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, M, N, K, float> acc_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    for (int i = 0; i < k; i += K) {
        wmma::load_matrix_sync(a_frag, A + warpM * k + i, k);
        wmma::load_matrix_sync(b_frag, B + i * n + warpN, n);
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }

    wmma::store_matrix_sync(C + warpM * n + warpN, acc_frag, n, wmma::mem_row_major);
}

int main(int argc, char** argv) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_file_path>" << std::endl;
        return 1;
    }
    const std::string filePath = argv[1];
    size_t m, n, k;
    parseDimensions(filePath, m, n, k);

    const size_t sizeA = m * n * sizeof(__half);
    const size_t sizeB = n * k * sizeof(__half);
    const size_t sizeC = m * k * sizeof(float);

    auto *h_A = static_cast<__half*>(malloc(sizeA));
    auto *h_B = static_cast<__half*>(malloc(sizeB));
    auto *h_C = static_cast<float*>(malloc(sizeC));

    loadHalfMatricesFromFileArray(filePath, h_A, m * n, h_B, n * k);

    __half *d_A, *d_B;
    float *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(WMMA_N, WMMA_N);
    dim3 blocksPerGrid((n + N * threadsPerBlock.x - 1) / (N * threadsPerBlock.x),
                       (m + M * threadsPerBlock.y - 1) / (M * threadsPerBlock.y));

    matrixMultiplyWMMA<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k);

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}
