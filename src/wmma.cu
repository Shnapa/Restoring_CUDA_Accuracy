#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <sstream>
#include <regex>

using namespace nvcuda;

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

void parseDimensionsFromFilename(const std::string& filename, size_t& m, size_t& k, size_t& n) {
    std::regex pattern(".*_(\\d+)x(\\d+)x(\\d+)\\.txt");
    std::smatch match;
    if (std::regex_match(filename, match, pattern)) {
        m = std::stoi(match[1]);
        k = std::stoi(match[2]);
        n = std::stoi(match[3]);
    } else {
        throw std::invalid_argument("Filename does not match expected format: " + filename);
    }
}

int loadHalfMatricesFromFileArray(const std::string &filePath, __half* A, size_t A_elements, __half* B, size_t B_elements) {
    std::ifstream file(filePath);
    if (!file.is_open()) return -1;

    std::string line;
    size_t count = 0;

    // Read matrix A
    while (count < A_elements && std::getline(file, line)) {
        std::istringstream iss(line);
        float val;
        while (iss >> val && count < A_elements) {
            A[count++] = __float2half(val);
        }
    }

    count = 0;
    while (count < B_elements && std::getline(file, line)) {
        std::istringstream iss(line);
        float val;
        while (iss >> val && count < B_elements) {
            B[count++] = __float2half(val);
        }
    }

    return 0;
}

__global__ void matrixMultiplyWMMA(const __half *A, const __half *B, float *C, int m, int n, int k) {
    int warpM = (blockIdx.y * blockDim.y + threadIdx.y);
    int warpN = (blockIdx.x * blockDim.x + threadIdx.x);

    if ((warpM + 1) * WMMA_M > m || (warpN + 1) * WMMA_N > n) return;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    wmma::fill_fragment(c_frag, 0.0f);

    for (int i = 0; i < k; i += WMMA_K) {
        if (i + WMMA_K <= k) {
            wmma::load_matrix_sync(a_frag, A + warpM * WMMA_M * k + i, k);
            wmma::load_matrix_sync(b_frag, B + i * n + warpN * WMMA_N, n);
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        }
    }

    wmma::store_matrix_sync(C + warpM * WMMA_M * n + warpN * WMMA_N, c_frag, n, wmma::mem_row_major);
}

int main(int argc, char** argv) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_file_path>" << std::endl;
        return 1;
    }

    const std::string filePath = argv[1];
    size_t m, k, n;
    try {
        parseDimensionsFromFilename(filePath, m, k, n);
    } catch (const std::invalid_argument& e) {
        std::cerr << e.what() << std::endl;
        return 1;
    }

    if (m % WMMA_M != 0 || k % WMMA_K != 0 || n % WMMA_N != 0) {
        std::cerr << "Error: WMMA requires matrix sizes to be multiples of 16." << std::endl;
        return 1;
    }

    const size_t sizeA = m * k * sizeof(__half);
    const size_t sizeB = k * n * sizeof(__half);
    const size_t sizeC = m * n * sizeof(float);

    auto *h_A = static_cast<__half*>(malloc(sizeA));
    auto *h_B = static_cast<__half*>(malloc(sizeB));
    auto *h_C = static_cast<float*>(malloc(sizeC));

    if (loadHalfMatricesFromFileArray(filePath, h_A, m * k, h_B, k * n) != 0) {
        std::cerr << "Failed to load matrices from file.\n";
        return 2;
    }

    __half *d_A, *d_B;
    float *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(2, 2);
    dim3 blocksPerGrid(n / WMMA_N / threadsPerBlock.x, m / WMMA_M / threadsPerBlock.y);

    matrixMultiplyWMMA<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    std::cout << "Result matrix C (" << m << "x" << n << "):\n";
    for (size_t i = 0; i < m; ++i) {
        for (size_t j = 0; j < n; ++j) {
            std::cout << h_C[i * n + j] << " ";
        }
        std::cout << "\n";
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
