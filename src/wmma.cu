#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <sstream>
#include <regex>

using namespace nvcuda;

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

void parseDimensionsFromFilename(const std::string& filename, size_t& m, size_t& k, size_t& n) {
    std::regex pattern(".*_(\\d+)x(\\d+)x(\\d+)\\.txt");
    std::smatch match;
    if (std::regex_match(filename, match, pattern)) {
        m = std::stoi(match[1]);
        k = std::stoi(match[2]);
        n = std::stoi(match[3]);
    } else {
        throw std::invalid_argument("Filename does not match expected format: " + filename);
    }
}

int loadHalfMatricesFromFileArray(const std::string &filePath, __half* A, size_t A_elements, __half* B, size_t B_elements) {
    std::ifstream file(filePath);
    if (!file.is_open()) return -1;

    std::string line;
    size_t count = 0;

    // Read matrix A
    while (count < A_elements && std::getline(file, line)) {
        std::istringstream iss(line);
        float val;
        while (iss >> val && count < A_elements) {
            A[count++] = __float2half(val);
        }
    }

    count = 0;
    while (count < B_elements && std::getline(file, line)) {
        std::istringstream iss(line);
        float val;
        while (iss >> val && count < B_elements) {
            B[count++] = __float2half(val);
        }
    }

    return 0;
}

__global__ void matrixMultiplyWMMA(const __half *A, const __half *B, float *C, int m, int n, int k) {
    int warpM = (blockIdx.y * blockDim.y + threadIdx.y);
    int warpN = (blockIdx.x * blockDim.x + threadIdx.x);

    if ((warpM * WMMA_M >= m) || (warpN * WMMA_N >= n)) return;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
    wmma::fill_fragment(acc_frag, 0.0f);

    for (int i = 0; i < k; i += WMMA_K) {
        if (i + WMMA_K <= k &&
            (warpM * WMMA_M + WMMA_M <= m) &&
            (warpN * WMMA_N + WMMA_N <= n)) {

            wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> a_frag;
            wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> b_frag;

            const __half *tileA = A + (warpM * WMMA_M) * k + i;
            const __half *tileB = B + i * n + (warpN * WMMA_N);

            wmma::load_matrix_sync(a_frag, tileA, k);
            wmma::load_matrix_sync(b_frag, tileB, n);

            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    if ((warpM * WMMA_M + WMMA_M <= m) && (warpN * WMMA_N + WMMA_N <= n)) {
        wmma::store_matrix_sync(C + (warpM * WMMA_M) * n + (warpN * WMMA_N), acc_frag, n, wmma::mem_row_major);
    }
}

int main(int argc, char** argv) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_file_path>" << std::endl;
        return 1;
    }

    const std::string filePath = argv[1];
    size_t m, k, n;
    try {
        parseDimensionsFromFilename(filePath, m, k, n);
    } catch (const std::invalid_argument& e) {
        std::cerr << e.what() << std::endl;
        return 1;
    }

    size_t padded_m = ((m + WMMA_M - 1) / WMMA_M) * WMMA_M;
    size_t padded_k = ((k + WMMA_K - 1) / WMMA_K) * WMMA_K;
    size_t padded_n = ((n + WMMA_N - 1) / WMMA_N) * WMMA_N;

    const size_t sizeA = padded_m * padded_k * sizeof(__half);
    const size_t sizeB = padded_k * padded_n * sizeof(__half);
    const size_t sizeC = padded_m * padded_n * sizeof(float);

    auto *h_A = static_cast<__half*>(calloc(padded_m * padded_k, sizeof(__half)));
    auto *h_B = static_cast<__half*>(calloc(padded_k * padded_n, sizeof(__half)));
    auto *h_C = static_cast<float*>(calloc(padded_m * padded_n, sizeof(float)));

    if (loadHalfMatricesFromFileArray(filePath, h_A, m * k, h_B, k * n) != 0) {
        std::cerr << "Failed to load matrices from file.\n";
        return 2;
    }

    __half *d_A, *d_B;
    float *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(2, 2);
    dim3 blocksPerGrid((padded_n / WMMA_N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (padded_m / WMMA_M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMultiplyWMMA<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, padded_m, padded_n, padded_k);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    std::cout << "Result matrix C (" << m << "x" << n << "):\n";
    for (size_t i = 0; i < m; ++i) {
        for (size_t j = 0; j < n; ++j) {
            std::cout << h_C[i * padded_n + j] << " ";
        }
        std::cout << "\n";
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
