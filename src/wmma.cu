
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <cstdio>

using namespace nvcuda;

#define M 16
#define N 16
#define K 16

#define M_TILES 4
#define N_TILES 4
#define K_TILES 4

#define M_TOTAL (M * M_TILES)
#define N_TOTAL (N * N_TILES)
#define K_TOTAL (K * K_TILES)

__global__ void matmul_wmma(half *A, half *B, float *D) {
    int warpId = threadIdx.x / 32;
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    int row = (blockRow * blockDim.y + warpId) * M;
    int col = blockCol * N;

    if (row >= M_TOTAL || col >= N_TOTAL) return;

    wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, M, N, K, float> acc_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    for (int tileK = 0; tileK < K_TOTAL; tileK += K) {
        wmma::load_matrix_sync(a_frag, A + row * K_TOTAL + tileK, K_TOTAL);
        wmma::load_matrix_sync(b_frag, B + tileK * N_TOTAL + col, N_TOTAL);
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }

    wmma::store_matrix_sync(D + row * N_TOTAL + col, acc_frag, N_TOTAL, wmma::mem_row_major);
}


int main() {
    half *A, *B;
    float *D;

    size_t sizeA = M * K * sizeof(half);
    size_t sizeB = K * N * sizeof(half);
    size_t sizeD = M * N * sizeof(float);

    hipMallocManaged(&A, sizeA);
    hipMallocManaged(&B, sizeB);
    hipMallocManaged(&D, sizeD);

    for (int i = 0; i < M_TOTAL * K_TOTAL; i++) A[i] = __float2half(1.0f);
    for (int i = 0; i < K_TOTAL * N_TOTAL; i++) B[i] = __float2half(1.0f);

    dim3 blockDim(32 * 4);
    dim3 gridDim(N_TILES, M_TILES);

    matmul_wmma<<<gridDim, blockDim>>>(A, B, D);
    hipDeviceSynchronize();


    printf("Result matrix D:\n");
    for (int i = 0; i < M_TOTAL; i++) {
        for (int j = 0; j < N_TOTAL; j++) {
            printf("%.1f ", D[i * N_TOTAL + j]);
        }
        printf("\n");
    }

    hipFree(A);
    hipFree(B);
    hipFree(D);

    return 0;
}
