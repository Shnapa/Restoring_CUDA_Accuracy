#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <sstream>
#include <regex>

using namespace nvcuda;

#define M 16
#define N 16
#define K 16

void parseDimensionsFromFilename(const std::string& filename, size_t& m, size_t& k, size_t& n) {
    std::regex pattern(".*_(\\d+)x(\\d+)x(\\d+)\\.txt");
    std::smatch match;
    if (std::regex_match(filename, match, pattern)) {
        m = std::stoi(match[1]);
        k = std::stoi(match[2]);
        n = std::stoi(match[3]);
    } else {
        throw std::invalid_argument("Filename does not match expected format: " + filename);
    }
}

int loadHalfMatricesFromFileArray(const std::string &filePath, __half* A, size_t A_elements, __half* B, size_t B_elements) {
    std::ifstream file(filePath);
    if (!file.is_open()) return -1;

    std::string line;
    size_t count = 0;

    // Read matrix A
    while (count < A_elements && std::getline(file, line)) {
        std::istringstream iss(line);
        float val;
        while (iss >> val && count < A_elements) {
            A[count++] = __float2half(val);
        }
    }

    count = 0;
    while (count < B_elements && std::getline(file, line)) {
        std::istringstream iss(line);
        float val;
        while (iss >> val && count < B_elements) {
            B[count++] = __float2half(val);
        }
    }

    return 0;
}

__global__ void matrixMultiplyWMMA(const __half *A, const __half *B, float *C, size_t m, size_t n, size_t k) {
    const size_t warpM = (blockIdx.y * blockDim.y + threadIdx.y) * M;
    const size_t warpN = (blockIdx.x * blockDim.x + threadIdx.x) * N;

    if (warpM >= m || warpN >= n) return;

    wmma::fragment<wmma::matrix_a, M, N, K, __half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, M, N, K, __half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, M, N, K, float> acc_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    for (int i = 0; i < k; i += K) {
        wmma::load_matrix_sync(a_frag, A + warpM * k + i, k);
        wmma::load_matrix_sync(b_frag, B + i * n + warpN, n);
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }

    wmma::store_matrix_sync(C + warpM * n + warpN, acc_frag, n, wmma::mem_row_major);
}

int main(int argc, char** argv) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_file_path>" << std::endl;
        return 1;
    }

    const std::string filePath = argv[1];
    size_t m, k, n;
    try {
        parseDimensionsFromFilename(filePath, m, k, n);
    } catch (const std::invalid_argument& e) {
        std::cerr << e.what() << std::endl;
        return 1;
    }

    const size_t sizeA = m * k * sizeof(__half);
    const size_t sizeB = k * n * sizeof(__half);
    const size_t sizeC = m * n * sizeof(float);

    auto *h_A = static_cast<__half*>(malloc(sizeA));
    auto *h_B = static_cast<__half*>(malloc(sizeB));
    auto *h_C = static_cast<float*>(malloc(sizeC));

    if (loadHalfMatricesFromFileArray(filePath, h_A, m * k, h_B, k * n) != 0) {
        std::cerr << "Failed to load matrices from file.\n";
        return 2;
    }

    __half *d_A, *d_B;
    float *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(2, 2);
    dim3 blocksPerGrid((n + N * threadsPerBlock.x - 1) / (N * threadsPerBlock.x),
                       (m + M * threadsPerBlock.y - 1) / (M * threadsPerBlock.y));

    matrixMultiplyWMMA<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    std::cout << "Result matrix C (" << m << "x" << n << "):\n";
    for (size_t i = 0; i < m; ++i) {
        for (size_t j = 0; j < n; ++j) {
            std::cout << h_C[i * n + j] << " ";
        }
        std::cout << "\n";
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
