#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <regex>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>




using namespace nvcuda;

#define WARP_SIZE 32
#define TILE_DIM 16

void parseDimensionsFromFilename(const std::string& filename, int& M, int& K, int& N) {
    std::regex pattern(".*_(\\d+)x(\\d+)x(\\d+)\\.txt");
    std::smatch match;
    if (std::regex_match(filename, match, pattern)) {
        M = std::stoi(match[1]);
        K = std::stoi(match[2]);
        N = std::stoi(match[3]);
    } else {
        throw std::runtime_error("Filename does not match expected format");
    }
}

void loadMatricesFromFile(const std::string& filename, std::vector<__half>& A, std::vector<__half>& B, std::vector<float>& C, int M, int K, int N) {
    std::ifstream infile(filename);
    if (!infile.is_open()) throw std::runtime_error("Could not open file");

    float val;
    int total_A = M * K;
    int total_B = K * N;
    int total_C = M * N;

    for (int i = 0; i < total_A; i++) {
        infile >> val;
        A[i] = __float2half(val);
    }
    for (int i = 0; i < total_B; i++) {
        infile >> val;
        B[i] = __float2half(val);
    }
    for (int i = 0; i < total_C; i++) {
        infile >> val;
        C[i] = val;
    }
}

__global__ void WMMAKernel(half *A, half *B, float *C, float *D, int M_GLOBAL, int N_GLOBAL, int K_GLOBAL) {
    int warpM = blockIdx.x;
    int warpN = blockIdx.y;

    if ((warpM + 1) * TILE_DIM > M_GLOBAL || (warpN + 1) * TILE_DIM > N_GLOBAL)
        return;

    wmma::fragment<wmma::matrix_a, TILE_DIM, TILE_DIM, TILE_DIM, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, TILE_DIM, TILE_DIM, TILE_DIM, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, TILE_DIM, TILE_DIM, TILE_DIM, float> acc_frag;
    wmma::fragment<wmma::accumulator, TILE_DIM, TILE_DIM, TILE_DIM, float> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    for (int i = 0; i < K_GLOBAL; i += TILE_DIM) {
        if ((i + TILE_DIM) <= K_GLOBAL) {
            const half *tileA = A + warpM * TILE_DIM * K_GLOBAL + i;
            const half *tileB = B + i * N_GLOBAL + warpN * TILE_DIM;
            wmma::load_matrix_sync(a_frag, tileA, K_GLOBAL);
            wmma::load_matrix_sync(b_frag, tileB, N_GLOBAL);
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    float *tileC = C + warpM * TILE_DIM * N_GLOBAL + warpN * TILE_DIM;
    wmma::load_matrix_sync(c_frag, tileC, N_GLOBAL, wmma::mem_row_major);

    for (int i = 0; i < acc_frag.num_elements; ++i)
        c_frag.x[i] += acc_frag.x[i];

    float *tileD = D + warpM * TILE_DIM * N_GLOBAL + warpN * TILE_DIM;
    wmma::store_matrix_sync(tileD, c_frag, N_GLOBAL, wmma::mem_row_major);
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        printf("Usage: ./main matrix_MxKxN.txt\n");
        return 1;
    }

    std::string filename = argv[1];
    int M, K, N;
    parseDimensionsFromFilename(filename, M, K, N);

    if (M < TILE_DIM || K < TILE_DIM || N < TILE_DIM) {
        printf("Matrix dimensions must be at least 16x16x16 for WMMA\n");
        return 0;
    }

    size_t size_A = M * K;
    size_t size_B = K * N;
    size_t size_C = M * N;

    std::vector<__half> h_A(size_A);
    std::vector<__half> h_B(size_B);
    std::vector<float> h_C(size_C);

    loadMatricesFromFile(filename, h_A, h_B, h_C, M, K, N);

    half *d_A, *d_B;
    float *d_C, *d_D;
    hipMalloc(&d_A, size_A * sizeof(half));
    hipMalloc(&d_B, size_B * sizeof(half));
    hipMalloc(&d_C, size_C * sizeof(float));
    hipMalloc(&d_D, size_C * sizeof(float));

    hipMemcpy(d_A, h_A.data(), size_A * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size_B * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C.data(), size_C * sizeof(float), hipMemcpyHostToDevice);

    dim3 threads(WARP_SIZE, 1); // 1 warp per thread block
    dim3 blocks(M / TILE_DIM, N / TILE_DIM);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    WMMAKernel<<<blocks, threads>>>(d_A, d_B, d_C, d_D, M, N, K);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("GPU execution time: %.3f ms\n", ms);
    printf("TFLOPS: %.2f\n", ((double)M * N * K * 2) / (ms * 1e6));

    // Copy result back to host and print
    std::vector<float> h_D(size_C);
    hipMemcpy(h_D.data(), d_D, size_C * sizeof(float), hipMemcpyDeviceToHost);

    printf("Result matrix D = A*B + C (%dx%d):\n", M, N);
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%.3f ", h_D[i * N + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    return 0;
}
