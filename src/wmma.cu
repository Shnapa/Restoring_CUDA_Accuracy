#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdlib>
#include "matrixParser.h"
#include <benchmark/benchmark.h>

using namespace nvcuda;

#define M 16
#define N 16
#define K 16

__global__ void matrixMultiplyWMMA(const half *A, const half *B, float *C, size_t m, size_t n, size_t k) {
    int warpM = (blockIdx.y * blockDim.y + threadIdx.y) * M;
    int warpN = (blockIdx.x * blockDim.x + threadIdx.x) * N;

    if (warpM >= m || warpN >= n) return;

    wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, M, N, K, float> acc_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    for (int i = 0; i < k; i += K) {
        wmma::load_matrix_sync(a_frag, A + warpM * k + i, k);
        wmma::load_matrix_sync(b_frag, B + i * n + warpN, n);
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }

    wmma::store_matrix_sync(C + warpM * n + warpN, acc_frag, n, wmma::mem_row_major);
}

static void BM_RunMultiplicationWMMA(benchmark::State &state, const std::string &filePath) {
    size_t m, n, k;
    parseDimensions(filePath, m, k, n);

    const size_t sizeA = m * k * sizeof(half);
    const size_t sizeB = k * n * sizeof(half);
    const size_t sizeC = m * n * sizeof(float);

    auto *h_A = static_cast<half*>(malloc(sizeA));
    auto *h_B = static_cast<half*>(malloc(sizeB));

    loadMatricesFromFileArrayHalf(filePath, h_A, m * k, h_B, k * n);

    half *d_A, *d_B;
    float *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(2, 2);
    dim3 blocksPerGrid((n + N * threadsPerBlock.x - 1) / (N * threadsPerBlock.x),
                       (m + M * threadsPerBlock.y - 1) / (M * threadsPerBlock.y));

    for (auto _ : state) {
        hipMemset(d_C, 0, sizeC);
        matrixMultiplyWMMA<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k);
        hipDeviceSynchronize();
        benchmark::ClobberMemory();
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
}

int main(int argc, char** argv) {
    for (const auto &filepath : filePaths) {
        benchmark::RegisterBenchmark(filepath.c_str(), [filepath](benchmark::State &state) {
            BM_RunMultiplicationWMMA(state, filepath);
        });
    }
    benchmark::Initialize(&argc, argv);
    benchmark::RunSpecifiedBenchmarks();
    return 0;
}
