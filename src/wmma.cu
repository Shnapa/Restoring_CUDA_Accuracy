
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <cstdio>

using namespace nvcuda;

#define M 16
#define N 16
#define K 16

__global__ void matmul_wmma(half *A, half *B, float *D) {
    wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, M, N, K, float> acc_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    wmma::load_matrix_sync(a_frag, A, K);
    wmma::load_matrix_sync(b_frag, B, N);

    wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    wmma::store_matrix_sync(D, acc_frag, N, wmma::mem_row_major);
}

int main() {
    half *A, *B;
    float *D;

    size_t sizeA = M * K * sizeof(half);
    size_t sizeB = K * N * sizeof(half);
    size_t sizeD = M * N * sizeof(float);

    hipMallocManaged(&A, sizeA);
    hipMallocManaged(&B, sizeB);
    hipMallocManaged(&D, sizeD);

    for (int i = 0; i < M * K; i++)
        A[i] = __float2half(1.0f); // всі елементи = 1.0

    for (int i = 0; i < K * N; i++)
        B[i] = __float2half(1.0f); // всі елементи = 1.0

    matmul_wmma<<<1, 32>>>(A, B, D);
    hipDeviceSynchronize();

    printf("Result matrix D:\n");
    for (int i = 0; i < 16; i++) {
        for (int j = 0; j < 16; j++) {
            printf("%.1f ", D[i * 16 + j]);
        }
        printf("\n");
    }

    hipFree(A);
    hipFree(B);
    hipFree(D);

    return 0;
}
