#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <sstream>
#include <regex>

using namespace nvcuda;

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

void parseDimensionsFromFilename(const std::string& filename, size_t& m, size_t& k, size_t& n) {
    std::regex pattern(".*_(\\d+)x(\\d+)x(\\d+)\\.txt");
    std::smatch match;
    if (std::regex_match(filename, match, pattern)) {
        m = std::stoi(match[1]);
        k = std::stoi(match[2]);
        n = std::stoi(match[3]);
    } else {
        throw std::invalid_argument("Filename does not match expected format: " + filename);
    }
}

int loadHalfMatricesFromFileArray(const std::string& filePath,
                                  __half* A, size_t m, size_t k, size_t padded_k,
                                  __half* B, size_t k_b, size_t n, size_t padded_n) {
    std::ifstream file(filePath);
    if (!file.is_open()) return -1;

    std::string line;
    size_t row = 0;

    // Load matrix A
    while (row < m && std::getline(file, line)) {
        std::istringstream iss(line);
        float val;
        size_t col = 0;
        while (iss >> val && col < k) {
            A[row * padded_k + col] = __float2half(val);
            ++col;
        }
        for (; col < padded_k; ++col) {
            A[row * padded_k + col] = __float2half(0.0f);
        }
        ++row;
    }
    for (; row < m; ++row) {
        for (size_t col = 0; col < padded_k; ++col) {
            A[row * padded_k + col] = __float2half(0.0f);
        }
    }

    // Load matrix B
    row = 0;
    while (row < k_b && std::getline(file, line)) {
        std::istringstream iss(line);
        float val;
        size_t col = 0;
        while (iss >> val && col < n) {
            B[row * padded_n + col] = __float2half(val);
            ++col;
        }
        for (; col < padded_n; ++col) {
            B[row * padded_n + col] = __float2half(0.0f);
        }
        ++row;
    }
    for (; row < padded_k; ++row) {
        for (size_t col = 0; col < padded_n; ++col) {
            B[row * padded_n + col] = __float2half(0.0f);
        }
    }

    return 0;
}

__global__ void matrixMultiplyWMMA(const __half* A, const __half* B, float* C, int m, int n, int k, int orig_m, int orig_n) {
    int warpM = blockIdx.y * blockDim.y + threadIdx.y;
    int warpN = blockIdx.x * blockDim.x + threadIdx.x;

    if ((warpM + 1) * WMMA_M > m || (warpN + 1) * WMMA_N > n) return;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
    wmma::fill_fragment(acc_frag, 0.0f);

    for (int i = 0; i < k; i += WMMA_K) {
        if (i + WMMA_K <= k) {
            wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> a_frag;
            wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> b_frag;

            const __half* tileA = A + (warpM * WMMA_M) * k + i;
            const __half* tileB = B + i * n + (warpN * WMMA_N);

            wmma::load_matrix_sync(a_frag, tileA, k);
            wmma::load_matrix_sync(b_frag, tileB, n);
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    // Store only if within original bounds
    if (warpM * WMMA_M < orig_m && warpN * WMMA_N < orig_n) {
        wmma::store_matrix_sync(C + (warpM * WMMA_M) * orig_n + (warpN * WMMA_N), acc_frag, orig_n, wmma::mem_row_major);
    }
}

int main(int argc, char** argv) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_file_path>" << std::endl;
        return 1;
    }

    const std::string filePath = argv[1];
    size_t m, k, n;
    try {
        parseDimensionsFromFilename(filePath, m, k, n);
    } catch (const std::invalid_argument& e) {
        std::cerr << e.what() << std::endl;
        return 1;
    }

    size_t padded_m = ((m + WMMA_M - 1) / WMMA_M) * WMMA_M;
    size_t padded_k = ((k + WMMA_K - 1) / WMMA_K) * WMMA_K;
    size_t padded_n = ((n + WMMA_N - 1) / WMMA_N) * WMMA_N;

    size_t sizeA = padded_m * padded_k * sizeof(__half);
    size_t sizeB = padded_k * padded_n * sizeof(__half);
    size_t sizeC = padded_m * padded_n * sizeof(float);

    auto* h_A = static_cast<__half*>(calloc(padded_m * padded_k, sizeof(__half)));
    auto* h_B = static_cast<__half*>(calloc(padded_k * padded_n, sizeof(__half)));
    auto* h_C = static_cast<float*>(calloc(padded_m * padded_n, sizeof(float)));

    if (loadHalfMatricesFromFileArray(filePath, h_A, m, k, padded_k, h_B, k, n, padded_n) != 0) {
        std::cerr << "Failed to load matrices from file.\n";
        return 2;
    }

    __half* d_A;
    __half* d_B;
    float* d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);
    hipMemset(d_C, 0, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(2, 2);
    dim3 blocksPerGrid((padded_n + WMMA_N * threadsPerBlock.x - 1) / (WMMA_N * threadsPerBlock.x),
                       (padded_m + WMMA_M * threadsPerBlock.y - 1) / (WMMA_M * threadsPerBlock.y));

    matrixMultiplyWMMA<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, padded_m, padded_n, padded_k, m, n);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    std::cout << "Result matrix C (" << m << "x" << n << "):\n";
    for (size_t i = 0; i < m; ++i) {
        for (size_t j = 0; j < n; ++j) {
            std::cout << h_C[i * n + j] << " ";
        }
        std::cout << "\n";
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
