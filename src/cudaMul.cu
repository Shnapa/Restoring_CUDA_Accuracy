#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "matrixParser.h"
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void cudaMul(const float* A, const float* B, float* C, size_t m, size_t n, size_t k) {
    const size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m && col < k) {
        float sum = 0.0f;
        for (size_t i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

int main(const int argc, char** argv) {
    if(argc < 2) {
       std::cerr << "Usage: " << argv[0] << " <matrix_file_path>" << std::endl;
       return 1;
    }
    const std::string filePath = argv[1];

    size_t m, n, k;
    parseDimensions(filePath, m, n, k);
    const size_t A_elements = m * n, B_elements = n * k, C_elements = m * k;

    auto* h_A = static_cast<float*>(malloc(A_elements * sizeof(float)));
    auto* h_B = static_cast<float*>(malloc(B_elements * sizeof(float)));
    loadMatricesFromFileArray(filePath, h_A, A_elements, h_B, B_elements);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, A_elements * sizeof(float));
    hipMalloc(&d_B, B_elements * sizeof(float));
    hipMalloc(&d_C, C_elements * sizeof(float));

    hipMemcpy(d_A, h_A, A_elements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, B_elements * sizeof(float), hipMemcpyHostToDevice);

    free(h_A);
    free(h_B);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((k + TILE_SIZE - 1) / TILE_SIZE, (m + TILE_SIZE - 1) / TILE_SIZE);

    cudaMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k);
    hipDeviceSynchronize();

    auto* h_C = static_cast<float*>(malloc(C_elements * sizeof(float)));
    hipMemcpy(h_C, d_C, C_elements * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "CUDA multiplication complete." << std::endl;
    std::cout << "First element of result: " << h_C[0] << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_C);
    return 0;
}
