#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "matrixParser.h"
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void cudaMulOpt(const float* A, const float* B, float* C, size_t m, size_t n, size_t k) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    size_t row = blockIdx.y * TILE_SIZE + threadIdx.y;
    size_t col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0f;
    for (size_t t = 0; t < (n + TILE_SIZE - 1) / TILE_SIZE; t++) {
        if (row < m && t * TILE_SIZE + threadIdx.x < n)
            tileA[threadIdx.y][threadIdx.x] = A[row * n + t * TILE_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;
        if (col < k && t * TILE_SIZE + threadIdx.y < n)
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * k + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;
        __syncthreads();
        for (int i = 0; i < TILE_SIZE; i++) {
            sum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < m && col < k)
        C[row * k + col] = sum;
}
inline bool compareFloats(float a, float b, float epsilon = 1e-2f) {
    float diff = fabsf(a - b);
    float maxVal = fmaxf(fabsf(a), fabsf(b));
    return diff <= epsilon * maxVal;
}
void compare(const float* h_C, size_t m, size_t n, size_t k, const std::string& filePath) {
    size_t A_elements = m * n;
    size_t B_elements = n * k;

    // Allocate and reload host matrices A and B
    auto* A = static_cast<float*>(malloc(A_elements * sizeof(float)));
    auto* B = static_cast<float*>(malloc(B_elements * sizeof(float)));
    loadMatricesFromFileArray(filePath, A, A_elements, B, B_elements);

    // Allocate memory for CPU result
    auto* C_cpu = static_cast<float*>(malloc(m * k * sizeof(float)));

    // Brute-force matrix multiplication
    for (size_t i = 0; i < m; ++i) {
        for (size_t j = 0; j < k; ++j) {
            float sum = 0.0f;
            for (size_t l = 0; l < n; ++l) {
                sum += A[i * n + l] * B[l * k + j];
            }
            C_cpu[i * k + j] = sum;
        }
    }

    // Compare CPU result with GPU result using epsilon for floating point comparison
    bool match = true;
    const float epsilon = 1e-5;
    for (size_t i = 0; i < m * k; ++i) {
        if (!compareFloats(C_cpu[i], h_C[i], epsilon)) {
            std::cerr << "Mismatch at index " << i << ": CPU = " << C_cpu[i]
                      << ", GPU = " << h_C[i] << std::endl;
            match = false;
            break;
        }
    }

    if (match) {
        std::cout << "Verification passed: CPU and GPU results match." << std::endl;
    } else {
        std::cout << "Verification failed: CPU and GPU results do not match." << std::endl;
    }

    free(A);
    free(B);
    free(C_cpu);
}

int main(const int argc, char** argv) {
    if(argc < 2) {
       std::cerr << "Usage: " << argv[0] << " <matrix_file_path>" << std::endl;
       return 1;
    }
    const std::string filePath = argv[1];

    size_t m, n, k;
    parseDimensions(filePath, m, n, k);
    const size_t A_elements = m * n, B_elements = n * k, C_elements = m * k;

    auto* h_A = static_cast<float*>(malloc(A_elements * sizeof(float)));
    auto* h_B = static_cast<float*>(malloc(B_elements * sizeof(float)));
    loadMatricesFromFileArray(filePath, h_A, A_elements, h_B, B_elements);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, A_elements * sizeof(float));
    hipMalloc(&d_B, B_elements * sizeof(float));
    hipMalloc(&d_C, C_elements * sizeof(float));

    hipMemcpy(d_A, h_A, A_elements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, B_elements * sizeof(float), hipMemcpyHostToDevice);

    free(h_A);
    free(h_B);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((k + TILE_SIZE - 1) / TILE_SIZE, (m + TILE_SIZE - 1) / TILE_SIZE);

    cudaMulOpt<<<blocks, threads>>>(d_A, d_B, d_C, m, n, k);
    hipDeviceSynchronize();

    auto* h_C = static_cast<float*>(malloc(C_elements * sizeof(float)));
    hipMemcpy(h_C, d_C, C_elements * sizeof(float), hipMemcpyDeviceToHost);
    compare(h_C, m, n, k, filePath);
    std::cout << "Optimized CUDA multiplication complete." << std::endl;
    std::cout << "First element of result: " << h_C[0] << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_C);
    return 0;
}
