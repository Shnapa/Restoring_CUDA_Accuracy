#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "matrixParser.h"
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void cudaMulOpt(const float* A, const float* B, float* C, size_t m, size_t n, size_t k) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    size_t row = blockIdx.y * TILE_SIZE + threadIdx.y;
    size_t col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0f;
    for (size_t t = 0; t < (n + TILE_SIZE - 1) / TILE_SIZE; t++) {
        if (row < m && t * TILE_SIZE + threadIdx.x < n)
            tileA[threadIdx.y][threadIdx.x] = A[row * n + t * TILE_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;
        if (col < k && t * TILE_SIZE + threadIdx.y < n)
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * k + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;
        __syncthreads();
        for (int i = 0; i < TILE_SIZE; i++) {
            sum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < m && col < k)
        C[row * k + col] = sum;
}

int main(const int argc, char** argv) {
    if(argc < 2) {
       std::cerr << "Usage: " << argv[0] << " <matrix_file_path>" << std::endl;
       return 1;
    }
    const std::string filePath = argv[1];

    size_t m, n, k;
    parseDimensions(filePath, m, n, k);
    const size_t A_elements = m * n, B_elements = n * k, C_elements = m * k;

    auto* h_A = static_cast<float*>(malloc(A_elements * sizeof(float)));
    auto* h_B = static_cast<float*>(malloc(B_elements * sizeof(float)));
    loadMatricesFromFileArray(filePath, h_A, A_elements, h_B, B_elements);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, A_elements * sizeof(float));
    hipMalloc(&d_B, B_elements * sizeof(float));
    hipMalloc(&d_C, C_elements * sizeof(float));

    hipMemcpy(d_A, h_A, A_elements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, B_elements * sizeof(float), hipMemcpyHostToDevice);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((k + TILE_SIZE - 1) / TILE_SIZE, (m + TILE_SIZE - 1) / TILE_SIZE);

    cudaMulOpt<<<blocks, threads>>>(d_A, d_B, d_C, m, n, k);
    hipDeviceSynchronize();

    auto* h_C = static_cast<float*>(malloc(C_elements * sizeof(float)));
    hipMemcpy(h_C, d_C, C_elements * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Optimized CUDA multiplication complete." << std::endl;
    std::cout << "First element of result: " << h_C[0] << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}
