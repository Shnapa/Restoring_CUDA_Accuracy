#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "timeMeasurement.h"
#include <iostream>
#include "matrixParser.h"

#define TILE_SIZE 32

__global__ void matrixMultiplyTiled(float *A, float *B, float *C, int N) {
    // Спільна пам’ять для тайлів
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    // Індекси поточного потоку
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    // Кількість підматриць, які потрібно обробити
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Завантажуємо дані в спільну пам’ять (перевіряємо межі)
        if (row < N && (t * TILE_SIZE + threadIdx.x) < N)
            tileA[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && (t * TILE_SIZE + threadIdx.y) < N)
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads(); // Синхронізація потоків

        // Виконуємо множення для даного тайлу
        for (int i = 0; i < TILE_SIZE; i++)
            sum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];

        __syncthreads(); // Очікуємо завершення перед наступною ітерацією
    }

    // Записуємо результат у глобальну пам’ять
    if (row < N && col < N)
        C[row * N + col] = sum;
}

void multiplyMatrices(float *h_A, float *h_B, float *h_C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    matrixMultiplyTiled<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(int argc, char* argv[]){
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <matrix_A_file> <matrix_B_file>" << std::endl;
        return 1;
    }
    std::string A_file = argv[1];
    std::string B_file = argv[2];

    MatrixData matA = parseMatrix(A_file);
    MatrixData matB = parseMatrix(B_file);

    if (matA.dim2 != matB.dim1) {
        std::cerr << "Dimension mismatch: A columns (" << matA.dim2
                  << ") != B rows (" << matB.dim1 << ")" << std::endl;
        return 1;
    }

    int n = static_cast<int>(matA.dim1);
    int k = static_cast<int>(matA.dim2);
    int m = static_cast<int>(matB.dim2);

    std::vector<double>& h_A = matA.data;
    std::vector<double>& h_B = matB.data;
    std::vector<double> h_C(n * m, 0.0);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeof(float) * n * k);
    hipMalloc(&d_B, sizeof(float) * k * m);
    hipMalloc(&d_C, sizeof(float) * n * m);

    hipMemcpy(d_A, h_A.data(), sizeof(float) * n * k, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), sizeof(float) * k * m, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    multiplyMatrices(d_A, d_B, d_C, n);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed_ms = 0.0f;
    hipEventElapsedTime(&elapsed_ms, start, stop);
    std::cout << "Elapsed time: " << std::fixed << elapsed_ms << " milisec" << std::endl;

    hipMemcpy(h_C.data(), d_C, n * m * sizeof(float), hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;

}