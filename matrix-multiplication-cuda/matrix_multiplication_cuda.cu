#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void matrixMultiplicationKernel(double *A, double *B, double *C, int n, int k, int m) {
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    double currentSum = 0.0;

    if (ROW < m && COL < n) {
        for (size_t i = 0; i < k; i++) {
            currentSum += A[ROW * k + i] * B[i * m + COL];
        }
    }

    C[ROW * m + COL] = currentSum;
}

void matrixMultiplication(double *A, double *B, double *C, int n, int k, int m) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((m + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, n, k, m);
}

int main() {
    int n = 2;
    int k = 2;
    int m = 2;

//    std::vector<double> h_A(n * k), h_B(k * m), h_C(n * m, 0.0);
    std::vector<double> h_A = {1, 1, 1, 1};
    std::vector<double> h_B = {2, 2, 2, 2};
    std::vector<double> h_C(n * m, 0.0);

    for (int i = 0; i < n * k; i++) h_A[i] = rand() % 10;
    for (int i = 0; i < k * m; i++) h_B[i] = rand() % 10;

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeof(double) * n * k);
    hipMalloc(&d_B, sizeof(double) * k * m);
    hipMalloc(&d_C, sizeof(double) * n * m);

    hipMemcpy(d_A, h_A.data(), sizeof(double) * n * k, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), sizeof(double) * k * m, hipMemcpyHostToDevice);

    matrixMultiplication(d_A, d_B, d_C, n, k, m);

    hipMemcpy(h_C.data(), d_C, n * m * sizeof(double), hipMemcpyDeviceToHost);

    std::cout << "Result matrix C:\n";
    for (int i = 0; i < n * m; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;

}