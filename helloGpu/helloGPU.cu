
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__ void helloGPU() {
    printf("Hello GPU\n");
}

int main() {
    helloGPU<<<1, 12>>>();

    hipDeviceSynchronize();
    return 0;
}